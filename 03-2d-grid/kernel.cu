#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "warper.h"
__device__ float distance(float2 x1, float2 x2){
    return sqrt(pow(x1.x - x2.x,2) + pow(x1.y - x2.y,2));
}
__global__ void distance_kernel(float2 *data_in, float *data_out, int w, int h){
    const int  x = blockIdx.x * blockDim.x + threadIdx.x;
    const int  y = blockIdx.y * blockDim.y + threadIdx.y;
    const int  i = x + y*w;
    if(x < w && y < h){
        float2 ref;
        ref.x = 0.0;
        ref.y = 0.0;
        data_out[i] = distance(data_in[i], ref);
    }
}

void run_kernel(float* h_in, float* h_out, int w, int h){
    float2 *d_in  = NULL;
    float *d_out = NULL;

    size_t in_size  = w*h*2*sizeof(float);
    size_t out_size = w*h*sizeof(float);
    
    dim3 block_dim(TX,TY);
    dim3 grid_dim((W + TX - 1)/TX,(H + TY - 1)/TY);
 
    //设备端分配内存
    hipMalloc((void**)&d_in, in_size);
    hipMalloc((void**)&d_out, out_size);

    //拷贝host数据到device
    hipMemcpy(d_in, h_in, in_size, hipMemcpyHostToDevice);

    distance_kernel<<<grid_dim,block_dim>>>(d_in, d_out, w, h);
    
    //拷贝device端计算结果到host
    hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}

