#include "hip/hip_runtime.h"
#include "warper.h"
__device__ float distance(float2 x1, float2 x2){
    return sqrt(pow(x1.x - x2.x,2) + pow(x1.y - x2.y,2));
}
__global__ void distance_kernel(float2 *data_in, float *data_out, int n){
    const int  i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        float2 ref;
        ref.x = 0.0;
        ref.y = 0.0;
        data_out[i] = distance(data_in[i], ref);
    }
}

void run_kernel(float* h_in, float* h_out, int n){
    float2 *d_in  = NULL;
    float *d_out = NULL;

    size_t in_size  = n*2*sizeof(float);
    size_t out_size = n*sizeof(float);
 
    //设备端分配内存
    hipMalloc((void**)&d_in, in_size);
    hipMalloc((void**)&d_out, out_size);

    //拷贝host数据到device
    hipMemcpy(d_in, h_in, in_size, hipMemcpyHostToDevice);

        
    distance_kernel<<<(N + TPB -1)/TPB,TPB>>>(d_in, d_out, n);
    //拷贝device端计算结果到host
    hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}

