
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <iostream>

__device__ float distance(float2 x1, float2 x2){
    return sqrt(pow(x1.x - x2.x,2) + pow(x1.y - x2.y,2));
}
__global__ void distance_kernel(float2 *data_in, float *data_out, int n){
    const int  i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        float2 ref;
        ref.x = 0.0;
        ref.y = 0.0;
        data_out[i] = distance(data_in[i], ref);
    }
}

void init_host_data(float2* h_in,int n){
    for(int i = 0; i< n;i++){
        h_in[i].x = (float)i /((n - 1) * M_PI * 100);
        h_in[i].y = sin(h_in[i].x);
    }
}

int main(){
    float *d_out = NULL;
    float2 *d_in  = NULL;
    float2 *h_in = NULL;
    float *h_out = NULL;
    int N           = 4096;
    int TPB         = 32;
    size_t in_size  = N*2*sizeof(float);
    size_t out_size = N*sizeof(float);
    h_in = (float2*)malloc(in_size);
    h_out = (float*)malloc(out_size);


    //设备端分配内存
    hipMalloc((void**)&d_in, in_size);
    hipMalloc((void**)&d_out, out_size);

    init_host_data(h_in, N);
    //拷贝host数据到device
    hipMemcpy(d_in, h_in, in_size, hipMemcpyHostToDevice);

    distance_kernel<<<(N + TPB -1)/TPB,TPB>>>(d_in, d_out, N);
    //拷贝device端计算结果到host
    hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
    
    for(int i = 0;i < N;i++){
        std::cout<<i<<":<"<<h_in[i].x<<","<<h_in[1].y<<">, dist:"<<h_out[i]<<std::endl;
    }
    free(h_in);
    free(h_out);
    return 0;
}
